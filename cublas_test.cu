#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void test(){
    printf("Hi Cuda World");
}

int main( int argc, char** argv )
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
