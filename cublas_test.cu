#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void test(){
    printf("Hi Cuda World");
}
void myprint(void);

int myprint()
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return printf("Hicuda");
}
